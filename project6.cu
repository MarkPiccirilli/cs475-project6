#include "hip/hip_runtime.h"
/********************************
* Title: Cuda Monte Carlo
* Name: Mark Piccirilli
* Course: CS 475
* Assignment: project 6
* Last Modified:
* Description:
* ************************************/

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		64000		// to make the timing more accurate
#endif

#ifndef NUMTRIES
#define NUMTRIES		100		// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

//ranges for random variables
const float XCMIN = 0.0;
const float XCMAX = 2.0;
const float YCMIN = 0.0;
const float YCMAX = 2.0;
const float RMIN = 0.5;
const float RMAX = 2.0;

// Monte Carlo Simulation

__global__  void MonteCarlo( float *xcs, float *ycs, float *rs, float* numHits )
{
	__shared__ float hits[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	//start out assuming its a hit.  If it is not a hit, adjust hits[tnum] to 0
	hits[tnum] = 1;

	float xc = xcs[gid];
	float yc = ycs[gid];
	float a = 2.;
	float b = -2.*(xc + yc);
	float c = xc*xc + yc*yc - r*r;
	float d = b*b - 4.*a*c;

	if(d < 0) {
		//laser misses circle
		hits[tnum] = 0;
	}

	d = sqrt(d);
	float t1 = (-b + d)/(2.*a);
	float t2 = (-b - d)/(2.*a);
	float tmin;
	//float tmin = t1 < t2 ? t1 : t2;
	if(t1 < t2) {
		tmin = t1;
	} 
	else {
		tmin = t2;
	}

	if(tmin < 0) {
		//circle engulfs laser
		hits[tnum] = 0;
	}

	float xcir = tmin;
	float ycir = tmin;

	//unitize normal vector
	float normalx = xcir - xc;
	float normaly = ycir - yc;
	float normal = sqrt(normalx*normalx + normaly*normaly);
	//unit vectors
	normalx /= normal;
	normaly /= normal;

	//unitized incoming vector
	float inx = xcir - 0.;
	float iny = ycir - 0.;
	float in = sqrt(inx*inx + iny*iny);

	//bounced vector
	float dot = inx*normalx + iny*normaly;
	float outx = inx - 2.*normalx*dot;
	float outy = iny - 2.*normaly*dot;

	float t = (0. - ycir)/outy;

	if(t < 0.) {
		//beam went up
		hits[tnum] = 0;
	}

	//add together all hits in block using reduction
	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			hits[tnum] += hits[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		numHits[wgNum] = hits[0];
}


// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:

	float * hxcs = new float [ NUMTRIALS ];
	float * hycs = new float [ NUMTRIALS ];
	float * hrc = new float [ NUMTRIALS ];
	int * hnumHits = new int [ NUMTRIALS/BLOCKSIZE ];

	for( int i = 0; i < NUMTRIALS; i++ )
	{
		hxcs[n] = Ranf(XCMIN, XCMAX);
		hycs[n] = Ranf(YCMIN, YCMAX);
		rs[n] = Ranf(RMIN, RMAX);
	}

	// allocate device memory:

	float *dxcs, *dycs, *drs, *dnumHits;

	dim3 dimsA( NUMTRIALS, 1, 1 );
	dim3 dimsB( NUMTRIALS, 1, 1 );
	dim3 dimsC( NUMTRIALS, 1, 1 );
	dim3 dimsD( SIZE/BLOCKSIZE, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dxcs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dycs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&drsx), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dnumHits), (SIZE/BLOCKSIZE)*sizeof(int) );
		checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dxcs, hxcs, NUMTRAILS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dycs, hycs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( drs, hrs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( NUMTRIALS / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

	for( int t = 0; t < NUMTRIALS; t++)
	{
	        MonteCarlo<<< grid, threads >>>( dxcs, dycs, drcs, dnumHits );
	}

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double trailsPerSecond = (float)NUMTRIALS * (float)NUMTRIES / secondsTotal;
	double megaTrailsPerSecond = trailsPerSecond / 1000000.;
	fprintf( stderr, "Number of trials = %10d, MegaTrials/Second = %10.2lf\n", NUMTRIALS, megaTrialsPerSecond );

	// copy result from the device to the host:

	status = hipMemcpy( hnumHits, dnumHits, (SIZE/BLOCKSIZE)*sizeof(int), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	// check the sum :

	int numHits = 0;
	for(int i = 0; i < NUMTRIALS/BLOCKSIZE; i++ )
	{
		//fprintf(stderr, "hnumHits[%6d] = %d\n", i, hnumHits[i]);
		numHits += hnumHits[i];
	}
	fprintf( stderr, "\nnumHits = %ld\n", numHits );
	
	//calculate frequency
	float frequency = numHits/NUMTRIALS;

	printf("frequency = %lf\n", frequency);

	// clean up memory:
	delete [ ] hxcs;
	delete [ ] hycs;
	delete [ ] hrs;
	delete [ ] hnumHits;

	status = hipFree( dxcs );
		checkCudaErrors( status );
	status = hipFree( dycs );
		checkCudaErrors( status );
	status = hipFree( drs );
		checkCudaErrors( status );
	status = hipFree( dnumHits );
		checkCudaErrors( status );


	return 0;
}

